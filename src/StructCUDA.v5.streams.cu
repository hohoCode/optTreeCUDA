#include "hip/hip_runtime.h"
//////////////////////////////Author: Hua He
//////////////////////////////OptTree CUDA Version 5 - Stream
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "StructCUDA.h"
#include ""

////Please make sure NumberOfThreads is a power of 2, for multithreads reduction purpose.
#define NumberOfThreads 64
////Please make sure the number of CUDA streams is an even number and >= 2, for better stream overlapping purpose.
#define STREAM_COUNT 32

extern "C" void kernel_wrapper (float* a, StructSimple* b, int* c, int d, int e, int f, int g);

__device__ float getLeafLoop(StructSimple* root, float* featureS) {
	StructSimple* item = root;
	while(item->left!=0||item->right!=0){
		float cc = *(featureS + item->fid);

		if( cc <= item->threshold) {
			if(item->left==0){
				break;
			}
			item = root + item->left;
		} else {
			if(item->right==0){
				break;
			}
			item = root + item->right;
		}
	}	
	return item->threshold;
}

__global__ void scoreAccumulator(StructSimple* tree, float* feature, int* nodeSizes, int numberOfInstances, int nbTrees, int numberOfFeatures, float* output){
	///The total thread number is numberOfInstances*nbTrees. Many threads will run CONCURRENTLY...
	int treeNo = blockIdx.y; 
	int cacheIndex = threadIdx.y * blockDim.x + threadIdx.x; 
	int instanceNo = blockIdx.x * blockDim.x + threadIdx.x; 
	__shared__ float out[NumberOfThreads];
	out[cacheIndex] = 0;
	__syncthreads();

	if(treeNo < nbTrees && instanceNo < numberOfInstances){
		int fStart = (numberOfFeatures)*instanceNo;
		int tStart = *(nodeSizes+treeNo);
		//////////////////////// Call the tree traversal code
		out[cacheIndex] = getLeafLoop(tree+tStart, feature+fStart);			
	}
	__syncthreads();
	int i = NumberOfThreads/2;
	while (i != 0) {
		if (cacheIndex < i)
			out[cacheIndex] += out[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}
	if(threadIdx.x == 0&&threadIdx.y==0){
		////////////////////Per block
		output[treeNo * gridDim.x + blockIdx.x] = out[0]; //Correct
	}
}


void kernel_wrapper(float* feature, StructSimple* tree, int* nodeSizes, int numberOfInstances, int nbTrees, int numberOfFeatures, int totalNodes){
	//////////////////CUDA Varibles Definition
	int* nodeSizes_cuda;
	StructSimple* tree_cuda;

	float *output[STREAM_COUNT];
	float *feature_stream[STREAM_COUNT];
	float *output_stream[STREAM_COUNT];
	hipStream_t stream[STREAM_COUNT];

	int OringinalNumberOfInstances = numberOfInstances;
	numberOfInstances = numberOfInstances/STREAM_COUNT;
	int SIZE = numberOfInstances * numberOfFeatures;
	int endSlot = OringinalNumberOfInstances - numberOfInstances * STREAM_COUNT + numberOfInstances;
	int endSize= endSlot * numberOfFeatures;

	/////////////////Timer
	hipEvent_t start_event, stop_event;
	float time;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);
	hipEventRecord(start_event, 0);
	
	/////////////////CUDA Dimension Definision
	dim3 dimBlock(NumberOfThreads, 1);
	int xx = (numberOfInstances + dimBlock.x - 1)/ dimBlock.x;
	int endxx = (endSlot + dimBlock.x - 1)/ dimBlock.x;
	int yy = (nbTrees + dimBlock.y -1)/ dimBlock.y;
	dim3 dimGrid1(endxx, yy);
	dim3 dimGrid(xx, yy);

	//////////////////CUDA Variable Initilization
	int ii = 0;
	for( ii =0; ii<STREAM_COUNT; ++ii ) {			
		hipStreamCreate(&stream[ii]);
		if(ii == STREAM_COUNT-1){	
			hipHostAlloc((void **) &output[ii], sizeof(float)* endxx * yy,  hipHostMallocDefault);
			hipMalloc((void**)&feature_stream[ii], sizeof(float) * endSize);
			hipMalloc((void**)&output_stream[ii], sizeof(float) * endxx * yy);		
		}else{
			hipHostAlloc((void **) &output[ii], sizeof(float)* xx * yy,  hipHostMallocDefault);
			hipMalloc((void**)&feature_stream[ii], sizeof(float) * SIZE);
			hipMalloc((void**)&output_stream[ii], sizeof(float) * xx * yy);		
		}			
	}
	hipMalloc((void**)&tree_cuda, sizeof(StructSimple) * totalNodes);
	hipMalloc((void**)&nodeSizes_cuda, sizeof(int) * nbTrees);

	//////////////////////////////////////////
	////////////Memory Move
	//////////////////////////////////////////
	hipMemcpy(tree_cuda, tree, sizeof(StructSimple) * totalNodes, hipMemcpyHostToDevice);
	hipMemcpy(nodeSizes_cuda, nodeSizes, sizeof(int) * nbTrees, hipMemcpyHostToDevice);	  
	ii = 0;

	///////////////////////////////CUDA WORK Start here 
	///////////////////////////////Using CUDA Stream ver 5
	///////////////////////////////Several streams working concurrently
	for( ii =0; ii<STREAM_COUNT; ii+=2 ){
		int next = ii+1;
		hipMemcpyAsync(feature_stream[ii], feature+ii*SIZE, sizeof(float)*SIZE, hipMemcpyHostToDevice, stream[ii]);		
		//////////////////////////The if condition is to deal with the final cuda block special situation.
		if(next == STREAM_COUNT-1){				
			hipMemcpyAsync(feature_stream[next], feature+next*SIZE, sizeof(float)*endSize, hipMemcpyHostToDevice, stream[next]);	
		}else{
			hipMemcpyAsync(feature_stream[next], feature+next*SIZE, sizeof(float)*SIZE, hipMemcpyHostToDevice, stream[next]);	
		}

		///////////////////////////REAL Work here - call real kernel functions
		scoreAccumulator<<<dimGrid, dimBlock, 0, stream[ii]>>>(tree_cuda, feature_stream[ii], nodeSizes_cuda, numberOfInstances, nbTrees, numberOfFeatures, output_stream[ii]);
		if(next == STREAM_COUNT-1){								
			scoreAccumulator<<<dimGrid1, dimBlock, 0, stream[next]>>>(tree_cuda, feature_stream[next], nodeSizes_cuda, endSlot, nbTrees, numberOfFeatures, output_stream[next]);
		}else{
			scoreAccumulator<<<dimGrid, dimBlock, 0, stream[next]>>>(tree_cuda, feature_stream[next], nodeSizes_cuda, numberOfInstances, nbTrees, numberOfFeatures, output_stream[next]);
		}

		//////////////////////////Error Checking
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit if any
			printf("CUDA error second: %s at Iteration: %d\n", hipGetErrorString(error), ii);
			exit(-1);
		}
		
		hipMemcpyAsync(output[ii], output_stream[ii], sizeof(float) * xx * yy, hipMemcpyDeviceToHost, stream[ii]);			  
		if(next == STREAM_COUNT-1){								
			hipMemcpyAsync(output[next], output_stream[next], sizeof(float) * endxx * yy, hipMemcpyDeviceToHost, stream[next]);
		}else{
			hipMemcpyAsync(output[next], output_stream[next], sizeof(float) * xx * yy, hipMemcpyDeviceToHost, stream[next]);
		}
	}
	/////////////////////////////////Waiting the work to be done in aync manner
	for( ii =0; ii<STREAM_COUNT; ++ii){
		hipStreamSynchronize( stream[ii] );
	}
	hipDeviceSynchronize();

	/////////////////////////////////Get the CUDA results out from device, get final sum
	double sum =0;
	int tindex = 0;
	for(ii=0; ii < STREAM_COUNT; ii++){
		if(ii == STREAM_COUNT-1){		
			for(tindex = 0; tindex < endxx * yy; tindex++) {
				sum += output[ii][tindex];				
			}	
		}else{
			for(tindex = 0; tindex < xx * yy; tindex++) {
				sum += output[ii][tindex];				
			}
		}
	}
	////////////////////////////////Timer Stop
	hipEventRecord(stop_event, 0);
	hipEventSynchronize(stop_event);
	hipEventElapsedTime(&time, start_event, stop_event);	

	float timeperinstance = time*1000000/(float)OringinalNumberOfInstances ;
	printf ("Total Time is %f ns, and Time/each instance: %f ns\n", time*1000000, timeperinstance);
	printf("Final Score is %.2f\n", sum);

	////////////////////////////////CUDA Memory Deallocation
	for( ii =0; ii<STREAM_COUNT; ++ii){
		hipFree(feature_stream[ii]); 
		hipFree(output_stream[ii]); 
		hipStreamDestroy(stream[ii]);
		hipHostFree(output[ii]);
	} 
	hipFree(tree_cuda);   
	hipFree(nodeSizes_cuda);   
}
